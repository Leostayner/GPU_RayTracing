#include "hip/hip_runtime.h"
#include <iostream>
#include <hiprand/hiprand_kernel.h>
#include <float.h>
#include "vec3.h"
#include "ray.h"
#include "hitable_list.h"
#include "sphere.h"
#include "camera.h"
#include "material.h"

//Alterar
__global__ void create_world(hitable **d_list, hitable **d_world, camera **d_camera) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        d_list[0] = new sphere(vec3(0,0,-1), 0.5, new lambertian(vec3(0.8, 0.3, 0.3)));
        d_list[1] = new sphere(vec3(0,-100.5,-1), 100, new lambertian(vec3(0.8, 0.8, 0.0)));
        d_list[2] = new sphere(vec3(1,0,-1), 0.5, new metal(vec3(0.8, 0.6, 0.2), 1.0));
        d_list[3] = new sphere(vec3(-1,0,-1), 0.5, new metal(vec3(0.8, 0.8, 0.8), 0.3));
        *d_world  = new hitable_list(d_list,4);
        *d_camera = new camera();
    }
}

//Alterar
__global__ void free_world(hitable **d_list, hitable **d_world, camera **d_camera) {
    for(int i=0; i < 4; i++) {
        delete ((sphere *)d_list[i])->mat_ptr;
        delete d_list[i];
    }
    delete *d_world;
    delete *d_camera;
}

__device__ vec3 color(const ray& r, hitable **world, hiprandState *rand_state) {
    ray cur_ray = r;
    vec3 cur_attenuation = vec3(1.0,1.0,1.0);
    for(int i = 0; i < 50; i++) {
        hit_record rec;
        if ((*world)->hit(cur_ray, 0.001f, FLT_MAX, rec)) {
            ray scattered;
            vec3 attenuation;
            if(rec.mat_ptr->scatter(cur_ray, rec, attenuation, scattered, rand_state)) {
                cur_attenuation *= attenuation;
                cur_ray = scattered;
            }
            else {
                return vec3(0.0,0.0,0.0);
            }
        }
        else {
            vec3 unit_direction = unit_vector(cur_ray.direction());
            float t = 0.5f*(unit_direction.y() + 1.0f);
            vec3 c = (1.0f-t)*vec3(1.0, 1.0, 1.0) + t*vec3(0.5, 0.7, 1.0);
            return cur_attenuation * c;
        }
    }
    return vec3(0.0,0.0,0.0);
}

__global__ void render(vec3 *img, int nx, int ny, int ns, hitable **world, camera **cam) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    
    if((i >= nx) || (j >= ny)) return;
    int pixel_index = j * nx + i;
    
    hiprandState state;
    hiprand_init((unsigned long long)clock64() + pixel_index, (unsigned long long)0, 0, &state);
    
    vec3 col(0,0,0);
    for(int s=0; s<ns; s++){
        float u = float(i + hiprand_uniform(&state)) / float(nx);
        float v = float(j + hiprand_uniform(&state)) / float(ny);
        ray r = (*cam)->get_ray(u, v);
        col += color(r, world, &state);
    }
    
    col /= float(ns);
    col[0] = sqrt(col[0]);
    col[1] = sqrt(col[1]);
    col[2] = sqrt(col[2]);
    img[pixel_index] = 255.99 * col;
}


int main() {
    int nx = 1200;
    int ny = 600;
    int ns = 100;
    int tx = 8;
    int ty = 8;
    
    dim3 blocks(nx/tx+1,ny/ty+1);
    dim3 threads(tx,ty);

    /**********/
    vec3 *img;
    hipMallocManaged((void **)&img, nx*ny*sizeof(vec3));

    hitable **list, **world; 
    hipMalloc((void **)&list, 2*sizeof(hitable *));
    hipMalloc((void **)&world, sizeof(hitable *));

    camera **cam;
    hipMalloc((void **)&cam, sizeof(camera *));

    
    /**********/
    create_world<<<1,1>>>(list, world, cam);
    hipDeviceSynchronize();
    
    render<<<blocks, threads>>>(img, nx, ny, ns, world, cam);
    hipDeviceSynchronize();

    /**********/
    std::cerr << "Rendering Image: " << nx << "x" << ny << std::endl;
    std::cout << "P3\n" << nx << " " << ny << "\n255\n";
    
    for (int j = ny-1; j >= 0; j--) {
        for (int i = 0; i < nx; i++) {
            size_t pixel_index = j*nx + i;
            int ir = int(img[pixel_index].r());
            int ig = int(img[pixel_index].g());
            int ib = int(img[pixel_index].b());
            std::cout << ir << " " << ig << " " << ib << "\n";
        }
    }

    /**********/
    hipDeviceSynchronize();
    free_world<<<1,1>>>(list, world, cam);
    void* freeList[4] = {cam, world, list, img};
    for(int i=0; i<4; i++) hipFree(freeList[i]);    
    hipDeviceReset();
}