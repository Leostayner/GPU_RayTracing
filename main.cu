#include "hip/hip_runtime.h"
#include <iostream>
#include "vec3.h"
#include "ray.h"

__device__ vec3 color(const ray& r) {
    vec3 unit_direction = unit_vector(r.direction());
    float t = 0.5f*(unit_direction.y() + 1.0f);
    return (1.0f-t)*vec3(1.0, 1.0, 1.0) + t*vec3(0.5, 0.7, 1.0);
}
__global__ void render(vec3 *img, int nx, int ny, vec3 lower_left_corner, vec3 horizontal, vec3 vertical, vec3 origin) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;

    if((i >= nx) || (j >= ny)) return;

    int pixel_index = j * nx + i;
    float u = float(i) / float(nx);
    float v = float(j) / float(ny);
    ray r(origin, lower_left_corner + u * horizontal + v * vertical);
    img[pixel_index] = color(r);
}


int main() {
    int nx = 1200;
    int ny = 600;
    int tx = 8;
    int ty = 8;

    std::cerr << "Rendering Image: " << nx << "x" << ny << std::endl;

    int num_pixels = nx*ny;
    size_t img_size = num_pixels*sizeof(vec3);

    vec3 *img;
    hipMallocManaged((void **)&img, img_size);

    dim3 blocks(nx/tx+1,ny/ty+1);
    dim3 threads(tx,ty);

    vec3 lower_left(-2.0, -1.0, -1.0);
    vec3 horizontal(4.0, 0.0, 0.0);
    vec3 vertical(0.0, 2.0, 0.0);
    vec3 origin(0.0, 0.0, 0.0);

    render<<<blocks, threads>>>(img, nx, ny, lower_left, horizontal, vertical, origin);
    hipDeviceSynchronize();
    
    std::cout << "P3\n" << nx << " " << ny << "\n255\n";
    
    for (int j = ny-1; j >= 0; j--) {
        for (int i = 0; i < nx; i++) {
            size_t pixel_index = j*nx + i;
            int ir = int(255.99*img[pixel_index].r());
            int ig = int(255.99*img[pixel_index].g());
            int ib = int(255.99*img[pixel_index].b());
            std::cout << ir << " " << ig << " " << ib << "\n";
        }
    }

    hipFree(img);
}