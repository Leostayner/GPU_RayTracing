#include "hip/hip_runtime.h"
#include <thrust/random/linear_congruential_engine.h>
#include <iostream>
#include "vec3.h"
#include "ray.h"
#include "hitable_list.h"
#include "sphere.h"
#include "camera.h"

__device__ vec3 random_in_unit_sphere() {
    vec3 p;
    thrust::minstd_rand rng1;

    do {
        p = 2.0f* vec3(rng1(), rng1(), rng1()) - vec3(1,1,1);
    } while (p.squared_length() >= 1.0f);
    return p;
}

__global__ void create_world(hitable **list, hitable **world, camera **cam) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        *(list)   = new sphere(vec3(0,0,-1), 0.5);
        *(list+1) = new sphere(vec3(0,-100.5,-1), 100);
        *world    = new hitable_list(list,2);
        *cam      = new camera();
    }
}

__global__ void free_world(hitable **list, hitable **world, camera **cam) {
    delete *(list);
    delete *(list+1);
    delete *world;
    delete *cam;
}

__device__ vec3 color(const ray& r, hitable **world) {
    hit_record rec;
    if ((*world)->hit(r, 0.001, MAXFLOAT, rec)) {
       vec3 target = rec.p + rec.normal + random_in_unit_sphere();
       return 0.5*color( ray(rec.p, target-rec.p), world);
    }
    else {
       vec3 unit_direction = unit_vector(r.direction());
       float t = 0.5*(unit_direction.y() + 1.0);
       return (1.0-t)*vec3(1.0, 1.0, 1.0) + t*vec3(0.5, 0.7, 1.0);
    }
 }

__global__ void render(vec3 *img, int nx, int ny, int ns, hitable **world, camera **cam) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;

    if((i >= nx) || (j >= ny)) return;
    
    int pixel_index = j * nx + i;
    vec3 col(0,0,0);
    for(int s=0; s<ns; s++){
        thrust::minstd_rand rng1;
        float u = float(i) + rng1()/ float(nx);
        float v = float(j) + rng1()/ float(ny);
        ray r = (*cam)->get_ray(u,v);
        col += color(r, world);
        
    }
    col /= float(ns);
    col[0] = sqrt(col[0]);
    col[1] = sqrt(col[1]);
    col[2] = sqrt(col[2]);
    img[pixel_index] = 255.99 * col;
}


int main() {
    int nx = 1200;
    int ny = 600;
    int ns = 100;
    int tx = 8;
    int ty = 8;
    
    dim3 blocks(nx/tx+1,ny/ty+1);
    dim3 threads(tx,ty);

    vec3 *img;
    hipMallocManaged((void **)&img, nx*ny*sizeof(vec3));

    hitable **list, **world; 
    hipMalloc((void **)&list, 2*sizeof(hitable *));
    hipMalloc((void **)&world, sizeof(hitable *));

    camera **cam;
    hipMalloc((void **)&cam, sizeof(camera *));

    create_world<<<1,1>>>(list, world, cam);
    hipDeviceSynchronize();
        
    render<<<blocks, threads>>>(img, nx, ny, ns, world, cam);
    hipDeviceSynchronize();

    std::cerr << "Rendering Image: " << nx << "x" << ny << std::endl;
    std::cout << "P3\n" << nx << " " << ny << "\n255\n";
    
    for (int j = ny-1; j >= 0; j--) {
        for (int i = 0; i < nx; i++) {
            size_t pixel_index = j*nx + i;
            int ir = int(img[pixel_index].r());
            int ig = int(img[pixel_index].g());
            int ib = int(img[pixel_index].b());
            std::cout << ir << " " << ig << " " << ib << "\n";
        }
    }

    hipFree(img);
    free_world<<<1,1>>>(list, world, cam);
}