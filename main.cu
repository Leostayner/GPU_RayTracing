#include "hip/hip_runtime.h"
#include <iostream>
#include <float.h>
#include "vec3.h"
#include "ray.h"
#include "hitable_list.h"
#include "sphere.h"
#include "camera.h"
#include <hiprand/hiprand_kernel.h>


__device__ vec3 random_in_unit_sphere(hiprandState *rand_state) {
    vec3 p;
    do {
        p = 2.0f*vec3(hiprand_uniform(rand_state), hiprand_uniform(rand_state), hiprand_uniform(rand_state)) - vec3(1,1,1);
    } while (p.squared_length() >= 1.0f);
    return p;
}

//Alterar
__global__ void create_world(hitable **list, hitable **world, camera **cam) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        *(list)   = new sphere(vec3(0,0,-1), 0.5);
        *(list+1) = new sphere(vec3(0,-100.5,-1), 100);
        *world    = new hitable_list(list,2);
        *cam      = new camera();
    }
}

//Alterar
__global__ void free_world(hitable **list, hitable **world, camera **cam) {
    delete *(list);
    delete *(list+1);
    delete *world;
    delete *cam;
}

__device__ vec3 color(const ray& r, hitable **world, hiprandState *rand_state) {
    ray cur_ray = r;
    float cur_attenuation = 1.0f;
    for(int i = 0; i < 50; i++) {
        hit_record rec;
        if ((*world)->hit(cur_ray, 0.001f, FLT_MAX, rec)) {
            vec3 target = rec.p + rec.normal + random_in_unit_sphere(rand_state);
            cur_attenuation *= 0.5f;
            cur_ray = ray(rec.p, target-rec.p);
        }
        else {
            vec3 unit_direction = unit_vector(cur_ray.direction());
            float t = 0.5f*(unit_direction.y() + 1.0f);
            vec3 c = (1.0f-t)*vec3(1.0, 1.0, 1.0) + t*vec3(0.5, 0.7, 1.0);
            return cur_attenuation * c;
        }
    }
    return vec3(0.0,0.0,0.0);
}

__global__ void render(vec3 *img, int nx, int ny, int ns, hitable **world, camera **cam) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    
    if((i >= nx) || (j >= ny)) return;
    int pixel_index = j * nx + i;
    
    hiprandState state;
    hiprand_init((unsigned long long)clock64() + pixel_index, (unsigned long long)0, 0, &state);
    
    vec3 col(0,0,0);
    for(int s=0; s<ns; s++){
        float u = float(i + hiprand_uniform(&state)) / float(nx);
        float v = float(j + hiprand_uniform(&state)) / float(ny);
        ray r = (*cam)->get_ray(u, v);
        col += color(r, world, &state);
    }
    
    col /= float(ns);
    col[0] = sqrt(col[0]);
    col[1] = sqrt(col[1]);
    col[2] = sqrt(col[2]);
    img[pixel_index] = 255.99 * col;
}


int main() {
    int nx = 1200;
    int ny = 600;
    int ns = 100;
    int tx = 8;
    int ty = 8;
    
    dim3 blocks(nx/tx+1,ny/ty+1);
    dim3 threads(tx,ty);

    /**********/
    vec3 *img;
    hipMallocManaged((void **)&img, nx*ny*sizeof(vec3));

    hitable **list, **world; 
    hipMalloc((void **)&list, 2*sizeof(hitable *));
    hipMalloc((void **)&world, sizeof(hitable *));

    camera **cam;
    hipMalloc((void **)&cam, sizeof(camera *));

    
    /**********/
    create_world<<<1,1>>>(list, world, cam);
    hipDeviceSynchronize();
    
    render<<<blocks, threads>>>(img, nx, ny, ns, world, cam);
    hipDeviceSynchronize();

    /**********/
    std::cerr << "Rendering Image: " << nx << "x" << ny << std::endl;
    std::cout << "P3\n" << nx << " " << ny << "\n255\n";
    
    for (int j = ny-1; j >= 0; j--) {
        for (int i = 0; i < nx; i++) {
            size_t pixel_index = j*nx + i;
            int ir = int(img[pixel_index].r());
            int ig = int(img[pixel_index].g());
            int ib = int(img[pixel_index].b());
            std::cout << ir << " " << ig << " " << ib << "\n";
        }
    }

    /**********/
    hipDeviceSynchronize();
    free_world<<<1,1>>>(list, world, cam);
    void* freeList[4] = {cam, world, list, img};
    for(int i=0; i<4; i++) hipFree(freeList[i]);    
    hipDeviceReset();
}